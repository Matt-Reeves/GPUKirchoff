#include "hip/hip_runtime.h"
#include <math.h>
#include <stdlib.h>
#include <stdio.h>
#include "useful.h"
#include "getCUDA.h"
#include "ark45ck.h"


//#define K1 a[i]
//#define K2 a[i+N]
//#define K3 a[i+2*N]
//#define K4 a[i+3*N]
//#define K5 a[i+4*N]
//#define K6 a[i+5*N]

#define R2 1.0
#define R1 0.1

__device__ inline void Positives(int tid,int j, int N,double L, double2 ri, double2 rj, double2* v,double* rmin) {
//Note L.x = D, L.y = q =  PI/D where D is the box size
// if ((ri.x == rj.x) && (ri.y = rj.y) ) printf("!! WARNING !!: ri (%d) == rj (%d), NaNs will follow. Aborting...\n",tid,j);
  double XIJ, YIJ;
  double k; 
  double sx, sy, cx, cy;
  XIJ = (ri.x - rj.x);
  YIJ = (ri.y - rj.y);
  if (tid < N/2){
   double rij;
    XIJ += (XIJ < -0.5*L) ? L:0.0;
    XIJ -= (XIJ >  0.5*L) ? L:0.0;
    YIJ += (YIJ < -0.5*L) ? L:0.0;
    YIJ -= (YIJ >  0.5*L) ? L:0.0;
    rij = XIJ*XIJ + YIJ*YIJ;
    if (rij < *rmin) *rmin = rij;
  }
  XIJ *= 2.0*PI/L;
  YIJ *= 2.0*PI/L;
  sincos(XIJ, &sx, &cx);
  sincos(YIJ, &sy, &cy);
  #pragma unroll
  for(k=-3.0; k<3.1; k+=1.0){ 
    (*v).x += (-sy)/(cosh(XIJ-2.0*PI*k)-cy);
    (*v).y += ( sx)/(cosh(YIJ-2.0*PI*k)-cx);
  }
}


__device__ inline void Negatives(int tid,int j,int N, double L, double2 ri, double2 rj, double2* v,double* rmin) {
 //if ((ri.x == rj.x) && (ri.y = rj.y) ) printf("!! WARNING !!: ri (%d) == rj (%d), NaNs will follow. Aborting...\n",tid,j);
  double XIJ, YIJ;
  double k; 
  double sx, sy, cx, cy;
  XIJ = (ri.x - rj.x);
  YIJ = (ri.y - rj.y);
  if (tid >= N/2 && tid < N){
    double rij;
    XIJ += (XIJ < -0.5*L) ? L:0.0;
    XIJ -= (XIJ >  0.5*L) ? L:0.0;
    YIJ += (YIJ < -0.5*L) ? L:0.0;
    YIJ -= (YIJ >  0.5*L) ? L:0.0;
    rij = XIJ*XIJ + YIJ*YIJ;
    if (rij < *rmin) *rmin = rij;
  }
  XIJ *= 2.0*PI/L;
  YIJ *= 2.0*PI/L;
  sincos(XIJ, &sx, &cx);
  sincos(YIJ, &sy, &cy);
  #pragma unroll
  for(k=-3.0; k<3.1; k+=1.0){ 
    (*v).x -=(-sy)/(cosh(XIJ-2.0*PI*k)-cy);
    (*v).y -=( sx)/(cosh(YIJ-2.0*PI*k)-cx);
  }
}


__global__ void Get_drdt(int N,double L,double dt,double gamma,double2* a,double2* rnew ){
   
  //Note that input argument "a" is now a pointer to EXACTLY the point in the array you want to dump values into,
  // no longer the first element of a. This removed the need to write a[i+(stage-1)*N].x etc....
  int i;
  double2 v;
  double local_gamma, rmin;
  extern __shared__ double2 smem[];
  double2 ri; 

  v.x = 0.0; v.y = 0.0; rmin = 1e10;
  i = threadIdx.x + blockIdx.x*blockDim.x;  
  /*Notice all threads do work here, even if they no longer have a vortex
  associated with them.  This is necessary because we still want them to load
  values into shared memory for the threads that still have work to do.*/
  ri = rnew[i];
  int p = 0; int tile; int idx;
  for (idx = threadIdx.x, tile = 0; tile < (N/2+blockDim.x-1)/blockDim.x; idx+= blockDim.x,tile++ ){
    smem[threadIdx.x] = rnew[idx];
    __syncthreads();
    for (int j=0; j<blockDim.x; j++){
      if (p == N/2) break; 
      if (i==p){ p++; continue;}
      Positives(i,p,N,L,ri, smem[j], &v,&rmin);
      p++;
    }
  __syncthreads();
  }
   if (p!= N/2) printf("!! WARNING !!: thread %d, p = %d\n",i,p);
  for (idx = threadIdx.x + N/2,tile = 0; tile < (N/2+blockDim.x-1)/blockDim.x; idx+= blockDim.x,tile++ ){
    smem[threadIdx.x] = rnew[idx];
    __syncthreads();
    for (int j=0; j<blockDim.x; j++){
      if (p == N) break; 
      if (i==p){ p++; continue; }
      Negatives(i,p,N,L,ri, smem[j], &v,&rmin);
      p++;
    }
  __syncthreads();
  }
   if (p!= N) printf("!! WARNING !!: thread %d, p = %d\n",i,p);
  //__syncthreads(); //Again, extra threads need to wait...

//  /*Failed attempt at negative viscosity .... */  
//  if (gamma < 0 ){ 
//    local_gamma = -gamma;
//    if (i < N/2 ) local_gamma = -local_gamma;
//
//    if (rmin <= R2) local_gamma *= -10;
//  }
//  else{ 

  //change the sign of gamma depending on thread index to avoid using kappai
   local_gamma =  -max(exp(log(gamma)*( (sqrt(rmin)-R1)/(R2-R1) )),gamma);
   
  if (i < N/2) local_gamma = -local_gamma;
//  }
  //only update if the thread wasn't calculating nonsense
  if (i<N){
    a[i].x = PI/L*(v.x + local_gamma*v.y)*dt;
    a[i].y = PI/L*(v.y - local_gamma*v.x)*dt;  
  }         
}


__global__ void doRungeKuttaStep1(int N,double2* r, double2* rnew, double2* k1){
  int i = threadIdx.x + blockIdx.x*blockDim.x;
  if (i<N){
    rnew[i].x = r[i].x + b21*k1[i].x; 
    rnew[i].y = r[i].y + b21*k1[i].y;
  }
} 
 
__global__ void doRungeKuttaStep2(int N,double2* r, double2* rnew, double2* k1, double2* k2){
  int i = threadIdx.x + blockIdx.x*blockDim.x;
  if (i<N){
    rnew[i].x = r[i].x + b31*k1[i].x + b32*k2[i].x; 
    rnew[i].y = r[i].y + b31*k1[i].y + b32*k2[i].y;
  }
}  
__global__ void doRungeKuttaStep3(int N,double2* r, double2* rnew, double2* k1, double2* k2, double2* k3){
  int i = threadIdx.x + blockIdx.x*blockDim.x;

  if (i<N){
    rnew[i].x = r[i].x + b41*k1[i].x + b42*k2[i].x + b43*k3[i].x; 
    rnew[i].y = r[i].y + b41*k1[i].y + b42*k2[i].y + b43*k3[i].y;
  }
}
__global__ void doRungeKuttaStep4(int N,double2* r, double2* rnew, double2* k1, double2* k2, double2* k3, double2* k4){
  int i = threadIdx.x + blockIdx.x*blockDim.x;

  if (i<N){
    rnew[i].x = r[i].x + b51*k1[i].x + b52*k2[i].x + b53*k3[i].x + b54*k4[i].x; 
    rnew[i].y = r[i].y + b51*k1[i].y + b52*k2[i].y + b53*k3[i].y + b54*k4[i].y;
  }
}
__global__ void doRungeKuttaStep5(int N,double2* r, double2* rnew, double2* k1, double2* k2, double2* k3, double2* k4, double2* k5){
  int i = threadIdx.x + blockIdx.x*blockDim.x;

  if (i<N){
    rnew[i].x = r[i].x + b61*k1[i].x + b62*k2[i].x + b63*k3[i].x + b64*k4[i].x + b65*k5[i].x; 
    rnew[i].y = r[i].y + b61*k1[i].y + b62*k2[i].y + b63*k3[i].y + b64*k4[i].y + b65*k5[i].y;
  }
}
__global__ void doRungeKuttaStep6(int N,double2* r, double2* rnew, double2* k1, double2* k2, double2* k3, double2* k4, double2* k5, double2* k6){
  int i = threadIdx.x + blockIdx.x*blockDim.x;
  if (i<N){
    //4th order...
    k2[i].x = r[i].x + c41*k1[i].x + c43*k3[i].x + c44*k4[i].x +c45*k5[i].x + c46*k6[i].x;
    k2[i].y = r[i].y + c41*k1[i].y + c43*k3[i].y + c44*k4[i].y +c45*k5[i].y + c46*k6[i].y;
    //5th order...
    k5[i].x = r[i].x + c51*k1[i].x + c53*k3[i].x + c54*k4[i].x + c56*k6[i].x;
    k5[i].y = r[i].y + c51*k1[i].y + c53*k3[i].y + c54*k4[i].y + c56*k6[i].y;
  }
}

//__global__ void doRungeKuttaStep1(int N,double2* r, double2* rnew, double2* a){
//  int i = threadIdx.x + blockIdx.x*blockDim.x;
//  if (i<N){
//    rnew[i].x = r[i].x + b21*K1.x; 
//    rnew[i].y = r[i].y + b21*K1.y;
//  }
//} 
// 
//__global__ void doRungeKuttaStep2(int N,double2* r, double2* rnew, double2* a){
//  int i = threadIdx.x + blockIdx.x*blockDim.x;
//  if (i<N){
//    rnew[i].x = r[i].x + b31*K1.x + b32*K2.x; 
//    rnew[i].y = r[i].y + b31*K1.y + b32*K2.y;
//  }
//}  
//__global__ void doRungeKuttaStep3(int N,double2* r, double2* rnew, double2* a){
//  int i = threadIdx.x + blockIdx.x*blockDim.x;
//  if (i<N){
//    rnew[i].x = r[i].x + b41*K1.x + b42*K2.x + b43*K3.x; 
//    rnew[i].y = r[i].y + b41*K1.y + b42*K2.y + b43*K3.y;
//  }
//}
//__global__ void doRungeKuttaStep4(int N,double2* r, double2* rnew, double2* a){
//  int i = threadIdx.x + blockIdx.x*blockDim.x;
//  if (i<N){
//    rnew[i].x = r[i].x + b51*K1.x + b52*K2.x + b53*K3.x + b54*K4.x; 
//    rnew[i].y = r[i].y + b51*K1.y + b52*K2.y + b53*K3.y + b54*K4.y;
//  }
//}
//__global__ void doRungeKuttaStep5(int N,double2* r, double2* rnew, double2* a){
//  int i = threadIdx.x + blockIdx.x*blockDim.x;
//  if (i<N){
//    rnew[i].x = r[i].x + b61*K1.x + b62*K2.x + b63*K3.x + b64*K4.x + b65*K5.x; 
//    rnew[i].y = r[i].y + b61*K1.y + b62*K2.y + b63*K3.y + b64*K4.y + b65*K5.y;
//  }
//}
//__global__ void doRungeKuttaStep6(int N,double2* r, double2* rnew, double2* a){
//  int i = threadIdx.x + blockIdx.x*blockDim.x;
//  if (i<N){
//    //4th order...
//    K2.x = r[i].x + c41*K1.x + c43*K3.x + c44*K4.x +c45*K5.x + c46*K6.x;
//    K2.y = r[i].y + c41*K1.y + c43*K3.y + c44*K4.y +c45*K5.y + c46*K6.y;
//    //5th order...
//    K5.x = r[i].x + c51*K1.x + c53*K3.x + c54*K4.x + c56*K6.x;
//    K5.y = r[i].y + c51*K1.y + c53*K3.y + c54*K4.y + c56*K6.y;
//  }
//}

